#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <fstream>
#include <sstream>
#include <string>
#include <cstring>
#include <cstdlib>
#include <cassert>
using namespace std;

typedef unsigned char byte;
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

void parse_metadata(char *file, string &csv, int &N, int &M) {
	ifstream metadata(file, ios::in);
	metadata >> csv >> N >> M;
	metadata.close();
}
//matrix[i][[j] = matrix[i*M+j]
void parse_line(string &line, byte *transaction, int row, int cols) {
	assert(transaction != NULL);
	stringstream ss(line);
	unsigned int value;
	while (ss >> value) {
		//transaction[value] = 1;
		transaction[row*cols + value] = 1;
		if (ss.peek() == ',')
			ss.ignore();
	}

}

void parse_transactions(string &file, byte *transactions, int N, int M) {
	ifstream csv(file, ios::in);
	string line;
	int i = 0;
	while (getline(csv, line)) {
		parse_line(line, transactions, i, M);
		i++;
	}
	csv.close();
}
__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void count1itemset(byte *transactions, int *N, int *M, int *counter) {
	int element = threadIdx.x;
	for (int i = 0; i < *N; i++) {
		for (int j = 0; j < *M; j++) {
			if (transactions[i*(*M)+j] == element)
				counter[element]++;
		}
	}
}

hipError_t get_1itemset(byte *transactions, int N, int M, float min_sup) {
	int *counter, *dev_N, *dev_M;
	int *ret = new int[M];
	byte *t;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto error;
	}
	cudaStatus = hipMalloc((void**)&counter, M*sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed! counter");
		goto error;
	}
	cudaStatus = hipMalloc((void**)&t, N*M*sizeof(byte));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed! transactions");
		goto error;
	}
	
	cudaStatus = hipMemcpy(t, transactions, N * M * sizeof(byte), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed! transactions");
		goto error;
	}
	hipMalloc((void **)&dev_N, sizeof(int));
	hipMalloc((void **)&dev_M, sizeof(int));
	hipMemcpy(dev_N, &N, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_M, &M, sizeof(int), hipMemcpyHostToDevice);
	cout << "before calling" << endl;
	count1itemset<<<1, M >>>(t, dev_N, dev_M, counter); //count?
	cout << "after calling" << endl;
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto error;
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto error;
	}
	cudaStatus = hipMemcpy(ret, counter, M * sizeof(int), hipMemcpyDeviceToHost);
	float temp;
	for (int i = 0; i < M; i++) {
		temp = ((float)counter[i]) / N;
		if (temp > min_sup)
			cout << i << endl;
	}
	error:
	delete[] ret;
	hipFree(counter);
	hipFree(t);
	return cudaStatus;
}
int main()
{
	/*
    const int arraySize = 5;
    const int a[arraySize] = { 1, 2, 3, 4, 5 };
    const int b[arraySize] = { 10, 20, 30, 40, 50 };
    int c[arraySize] = { 0 };

    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
	*/
	int N, M;
	string csv;
	float min_sup = 0.1;
	float minconf = 0.5;

	parse_metadata("input.txt", csv, N, M);
	M++;
	byte *transactions = new byte[N*M]();
	//int **transactions = new int*[N];
	
	parse_transactions(csv, transactions,N,M);
	get_1itemset(transactions, N, M, min_sup);

	delete[] transactions;
	return 0;
}



//suddivisione matrice?


// Helper function for using CUDA to add vectors in parallel.
/*
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
*/
